#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RANGE 11.79

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float *a, float *b, float *c, int n)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    
    while(id < n){
        c[id] += a[id] * b[id];
		id += gridDim.x * blockDim.x;
    }
            
    
}

/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	/*if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}*/
		
	n = atoi(argv[1]);
	//n = 1000000;
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++){
		temp[i] += a[i] * b[i];
	}
		//printf(" temp %f %f %f \n",temp[i],a[i],b[i]);
		
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	double CPU = (double)(end - start) / CLOCKS_PER_SEC;
    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	start = clock();
	
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/
	
	hipMalloc(&ad,n*sizeof(float)) ;
	hipMalloc(&bd,n*sizeof(float)) ;
	hipMalloc(&cd,n*sizeof(float)) ;

	hipMemcpy( ad, a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( bd, b, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( cd, c, n*sizeof(float), hipMemcpyHostToDevice);
	int blockSize = 500;
	int gridSize = 8;
	
    vecGPU<<<gridSize, blockSize>>>(ad, bd, cd, n);
	
	hipMemcpy( c, cd, n*sizeof(float), hipMemcpyDeviceToHost );
	hipFree(ad);
    hipFree(bd);
    hipFree(cd);
	
	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	double GPU = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Total time difference = %lf\n", GPU-CPU);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	//c[1000] = 1;
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if( abs(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		printf("Element %d in the result array does not match the sequential version %f %f\n", i,temp[i],c[i]);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/
