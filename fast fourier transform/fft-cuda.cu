#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <complex.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#ifndef M_PI
    #define M_PI 3.14159265358979323846
#endif

__device__ uint32_t revBits(uint32_t val){

    val = ((val & 0xaaaaaaaa) >> 1) | ((val & 0x55555555) << 1);
    val = ((val & 0xcccccccc) >> 2) | ((val & 0x33333333) << 2);
    val = ((val & 0xf0f0f0f0) >> 4) | ((val & 0x0f0f0f0f) << 4);
    val = ((val & 0xff00ff00) >> 8) | ((val & 0x00ff00ff) << 8);
    return (val >> 16) | (val << 16);
}


__global__ void fft_kernel(hipFloatComplex* input, hipFloatComplex* output, uint32_t N, int logN){

    uint32_t index = blockDim.x * blockIdx.x + threadIdx.x;

    uint32_t rev = revBits(2*index);
    rev = rev>>(32-logN);
    output[2*index] = input[rev]; 

    rev = revBits(2*index+1);
    rev = rev>>(32-logN);
    output[2*index+1] = input[rev]; 

    __syncthreads();

    for(int Findex = 1; Findex<=logN; Findex++){

        int size = 1<<Findex;
        int size1 = 1<<(Findex-1);

        int j = threadIdx.x / size1 * size;
        int k = threadIdx.x % size1;

        hipFloatComplex t1 = output[j+k];

        float real, imaginary;

        sincosf((float) -M_PI * k/size1, &imaginary, &real);
        hipFloatComplex twiddle = make_hipFloatComplex(real, imaginary);

        hipFloatComplex t2 = hipCmulf(twiddle, output[j+k+size1]);

        output[j+k] = hipCaddf(t1, t2);
        output[j+k+size1] = hipCsubf(t1, t2);

        __syncthreads();
    }
}


int main(int argc, char *argv[]){

    char* filename = argv[1];
    FILE* fp = fopen (filename, "r");
    if (fp == NULL) {
        printf ("Opening file failed. Please try again.\n");
        return 0;
    }

    uint32_t N;
    fscanf (fp, "%u\n", &N);

    hipFloatComplex* input = (hipFloatComplex*) malloc(N * sizeof(hipFloatComplex));
    hipFloatComplex* output = (hipFloatComplex*) malloc(N * sizeof(hipFloatComplex)); 
    
    for(uint32_t i = 0; i<N; i++){
        float a, b;
        fscanf (fp, "%f %f\n", &a, &b);
        input[i] = make_hipFloatComplex(a,b);
    }
    
    int logN = (int) log2f((float) N);

    hipFloatComplex* input_d;
    hipFloatComplex* output_d;

    hipMalloc((void**)&input_d, N * sizeof(hipFloatComplex));
    hipMalloc((void**)&output_d, N * sizeof(hipFloatComplex));

    hipMemcpy(input_d, input, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    
    int size = N/2;
    int block_size = min(size, 1024);
    dim3 block(block_size, 1);
    dim3 grid((size + block_size - 1) / block_size, 1);

    fft_kernel<<<grid, block>>>(input_d, output_d, N, logN);

    hipMemcpy(output, output_d, N*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    // for(int i=0; i<N; i++){
    //     printf("%f + i%f\n", hipCrealf(output[i]), hipCimagf(output[i]));
    // }

    hipFree(input_d);
    hipFree(output_d);

}